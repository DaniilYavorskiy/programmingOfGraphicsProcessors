﻿#pragma comment (lib, "cublas.lib")
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <time.h>
#include <iostream>

int const M = 1024;
int const N = 1024;

int main()
{
    hipblasStatus_t status;
    float tmr = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float* h_A = (float*)malloc(N * M * sizeof(float));
    float* h_B = (float*)malloc(N * M * sizeof(float));


    float* h_C = (float*)malloc(M * M * sizeof(float));


    for (int i = 0; i < N * M; i++) {
        h_A[i] = (float)(rand() % 10 + 1);
        h_B[i] = (float)(rand() % 10 + 1);

    }


    /*std::cout << "Matrix A:" << std::endl;
    for (int i = 0; i < N * M; i++) {
        std::cout << h_A[i] << " ";
        if ((i + 1) % N == 0) std::cout << std::endl;

    }
    std::cout << std::endl;
    std::cout << "Matrix B:" << std::endl;
    for (int i = 0; i < N * M; i++) {
        std::cout << h_B[i] << " ";
        if ((i + 1) % M == 0) std::cout << std::endl;
    }
    std::cout << std::endl;*/


    hipblasHandle_t handle;
    status = hipblasCreate(&handle);

    float* d_A, * d_B, * d_C;
    hipMalloc(
        (void**)&d_A,
        N * M * sizeof(float)
    );
    hipMalloc(
        (void**)&d_B,
        N * M * sizeof(float)
    );

    hipMalloc(
        (void**)&d_C,
        M * M * sizeof(float)
    );

    hipblasSetVector(
        N * M,
        sizeof(float),
        h_A,
        1,
        d_A,
        1
    );
    hipblasSetVector(

        N * M,
        sizeof(float),
        h_B,
        1,
        d_B,
        1
    );

    hipDeviceSynchronize();

    float a = 1; float b = 0;
    hipEventRecord(start, 0);
    hipblasSgemm(
        handle,
        HIPBLAS_OP_T,
        HIPBLAS_OP_T,
        M, // A, C строки 
        M, // B, C
        N,
        &a,
        d_A,
        N,
        d_B,
        M,
        &b,
        d_C,
        M
    );
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tmr, start, stop);
    std::cout << "cublas" << tmr << "microsecs" << std::endl;


    hipDeviceSynchronize();

    hipblasGetVector(
        M * M,
        sizeof(float),
        d_C,
        1,
        h_C,
        1
    );

    /*std::cout << "Transpose calculation results ((A * B) transpose):" << std::endl;*/

    /*for (int i = 0; i < M * M; i++) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % M == 0) std::cout << std::endl;
    }*/

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    hipblasDestroy(handle);

    return 0;
}

