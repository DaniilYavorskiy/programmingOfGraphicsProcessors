#include "hip/hip_runtime.h"
﻿//7.2
#include <>
#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/gather.h>
#include <thrust/transform.h>

#define THREADS_PER_BLOCK 7

__global__ void gFunc(int* A, int* B, int* C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= N) {
        return;
    }
    C[i] = A[i] * B[i];
}


int main(int argc, char** argv) {
    int N = 15;


    int* hA, * hB, * hC;
    int* A = (int*)calloc(N, sizeof(int));
    int* B = (int*)calloc(N, sizeof(int));
    int* C = (int*)calloc(N, sizeof(int));

    hipMalloc((void**)&hA, N * sizeof(int));
    hipMalloc((void**)&hB, N * sizeof(int));
    hipMalloc((void**)&hC, N * sizeof(int));



    srand(time(0));
    for (int i = 0; i < N; ++i) {
        A[i] = rand() % 100;
    }
    for (int i = 0; i < N; ++i) {
        B[i] = rand() % 100;
    }

    for (int i = 0; i < N; ++i) {
        printf("%4d ", A[i]);
    }
    printf("\n");
    hipMemcpy(hA, A, N * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < N; ++i) {
        printf("%4d ", B[i]);
    }
    printf("\n");


    hipMemcpy(hB, B, N * sizeof(int), hipMemcpyHostToDevice);
    float elapsedTime = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    gFunc << <dim3(THREADS_PER_BLOCK), dim3((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK) >> > (hA, hB, hC, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\ntime: %f\n", elapsedTime);

    hipMemcpy(C, hC, N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i) {
        printf("%4d ", C[i]);
    }

    printf("\n\n");




    hipFree(hA);
    hipFree(hB);
    hipFree(hC);

    thrust::host_vector<int> vA(A, A + N);
    thrust::host_vector<int> vB(B, B + N);
    thrust::host_vector<int> vC(N);

    for (int i = 0; i < N; ++i) {
        printf("%4d ", vA[i]);
    }
    printf("\n");
    for (int i = 0; i < N; ++i) {
        printf("%4d ", vB[i]);
    }

    thrust::device_vector<int> dA = vA;
    thrust::device_vector<int> dB = vB;
    thrust::device_vector<int> dC(N);


    hipEventRecord(start, 0);
    thrust::transform(dA.begin(), dA.end(), dB.begin(), dC.begin(), thrust::multiplies<int>());
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\n\ntime: %f\n", elapsedTime);
    vC = dC;
    for (int i = 0; i < N; ++i) {
        printf("%4d ", vC[i]);

    }
    printf("\n\n");
    free(A);
    free(B);
    free(C);

}


