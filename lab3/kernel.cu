#include "hip/hip_runtime.h"
﻿#include <cstdlib>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

__global__ void vectors_add(float arr1[], float arr2[])
{
	size_t i = threadIdx.x + blockDim.x * blockIdx.x;
	arr1[i] += arr2[i];
}

void print(int num) {
	long size = 1 << 20;
	float* arr1, * arr2, * res, * devarr1, * devarr2;
	long threads_per_block = num;
	long num_of_blocks = size / threads_per_block;
	arr1 = new float[size];
	arr2 = new float[size];
	res = new float[size];
	for (long i = 0; i < size; i++) {
		arr1[i] = (float)rand() / RAND_MAX;
		arr2[i] = (float)rand() / RAND_MAX;
	}
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((void**)&devarr1, size * sizeof(float));
	hipMalloc((void**)&devarr2, size * sizeof(float));
	hipMemcpy(devarr1, arr1, size * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(devarr2, arr2, size * sizeof(float),
		hipMemcpyHostToDevice);
	hipEventRecord(start, 0);
	vectors_add <<< dim3(num_of_blocks),
		dim3(threads_per_block) >> > (devarr1, devarr2);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("%f", time * 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
	hipFree(devarr1);
	hipFree(devarr2);
	delete[] arr1;
	delete[] arr2;
	delete[] res;
}
int main(int argc, char* argv[])
{
	int arr[11] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024};
	for (int i = 0; i < 11; i++) {
		print(arr[i]);
		std::cout << ' ' << arr[i] << std::endl;
	}
	return 0;
}

