﻿#pragma comment (lib, "cublas.lib") 
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <hipblas.h>
#include <cstdlib>
#include <chrono>
#include "hip/device_functions.h"


void printTable(float* m, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            printf("%6.2f ", m[i * size + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    srand(time(0));
    const int MAXSTEP = 4096;

    FILE* file = fopen("resultsCU.csv", "a+");

    float* A = new float[MAXSTEP * MAXSTEP];
    float* B = new float[MAXSTEP * MAXSTEP];
    float* C = new float[MAXSTEP * MAXSTEP];
    float* deviceA;
    hipMalloc(&deviceA, MAXSTEP * MAXSTEP * sizeof(float));
    float* deviceB;
    hipMalloc(&deviceB, MAXSTEP * MAXSTEP * sizeof(float));
    float* deviceC;
    hipMalloc(&deviceC, MAXSTEP * MAXSTEP * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    for (int i = 2; i <= MAXSTEP; i *= 2) {


        for (int j = 0; j < i * i; ++j) {
            A[j] = rand() / static_cast<float>(RAND_MAX);
            B[j] = rand() / static_cast<float>(RAND_MAX);
        }


        hipblasSetMatrix(i, i, sizeof(float), A, i, deviceA, i);
        //	 		cudaMemcpy(deviceA, A, i * i * sizeof(float), cudaMemcpyHostToDevice);		

        hipblasSetMatrix(i, i, sizeof(float), B, i, deviceB, i);
        //	 cudaMemcpy(deviceB, B, i * i * sizeof(float), cudaMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);


        const float alpha = 1.0f;
        const float beta = 0.0f;


        hipEventRecord(start, 0);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, i, i, i, &alpha, deviceA, i, deviceB, i, &beta, deviceC, i);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);


        float time;
        hipEventElapsedTime(&time, start, stop);



        //			cudaMemcpy(C, deviceC, i * i * sizeof(float), cudaMemcpyDeviceToHost);
        hipblasGetMatrix(i, i, sizeof(float), deviceC, i, C, i);

        //		printTable(A, i);
            //		printTable(B, i);
                //	printTable(C, i);


        if (i == 2) {
            fprintf(file, "SIZE;TIME;\n");
        }
        fprintf(file, "%d;%f;\n", i, time);


    }

    hipblasDestroy(handle);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
